#include <hip/hip_runtime.h>
#include <limits.h>
#include <stdio.h>


__device__ const char *conservativeGroup[] = {"NDEQ", "MILV", "FYW", "NEQK", "QHRK", "HY", "STA", "NHQK", "MILF"};
__device__ const char *semiConservativeGroup[] = {"SAG", "SGND", "NEQHRK", "ATV", "STPA", "NDEQHK", "HFY", "CSA", "STNK", "SNDEQK", "FVLIM"};

typedef struct
{
    int n;
    int k;
}nkTuple;


// calculate string length
__host__ __device__ int strlenDevice(const char *str)
{
    int i = 0;
    while (str[i] != '\0')
    {
        i++;
    }
    return i;
}

__device__ int inString(const char *str, char c)
{
    for (int i = 0; str[i] != '\0'; i++)
    {
        if (str[i] == c)
        {
            return 0;
        }
    }
    return -1;
}

__device__ int compareInGroup(char a, char b, const char *group[], int groupLength)
{
    for (int i = 0; i < 9; i++)
    {
        if (inString(group[i], a) && inString(group[i], b))
            return 0;
    }
    return -1;
}

__device__ char getSign(char a, char b)
{
    if (a == b)
    {
        return '*';
    }
    switch (compareInGroup(a, b, conservativeGroup, 9))
    {
    case 0:
        return ':';
    case -1:
        switch (compareInGroup(a, b, semiConservativeGroup, 11))
        {
        case 0:
            return '.';
        default: // case -1: not in semiConservativeGroup
            return ' ';
        }
    }
    return ' ';
}

__device__ int calculateScore(char *seq1, char *seq2, int w[], int offset)
{
    int starsCount = 0;
    int colonsCount = 0;
    int pointsCount = 0;
    int spacesCount = 0;
    for (int i = 0; i < strlenDevice(seq2); i++)
    {
        switch (getSign(seq1[i + offset], seq2[i]))
        {
        case '*':
            starsCount++;
            break;
        case ':':
            colonsCount++;
            break;
        case '.':
            pointsCount++;
            break;
        case ' ':
            spacesCount++;
            break;
        }
    }
    return w[0] * starsCount - w[1] * colonsCount - w[2] * pointsCount - w[3] * spacesCount;
}

__device__ void getNKFromNumber(nkTuple *tuple,int number, int lengthSeq2)
{
    tuple->n = 0;
    tuple->k = 1;
    for(int i = 0; i < number; i++)
    {
        if (tuple->k < lengthSeq2 - 1) // increment k
            (tuple->k)++;
        else // increment n and reset k
        {
            (tuple->n)++;
            tuple->k = (tuple->n) + 1;
        }
    }

}

__global__ void BestMutantANDOffset(nkTuple* nk, char* seq1, char* seq2, int* w, int* offset,int* maxScore)           // each block represent mutant, each thread represent offset.
{
    __shared__ int bestOffset;
    __shared__ int bestScore;
    __shared__ char* seq2Shared;
    __shared__ nkTuple myTuple;
    *maxScore = INT_MIN;

    if(threadIdx.x == 0)
    {
        bestOffset = 0;
        bestScore = INT_MIN;
        int len =  strlenDevice(seq2);
        
        seq2Shared = (char*)malloc(len-1);
        
        seq2Shared[strlenDevice(seq2) - 2] = '\0';
                
        getNKFromNumber(&myTuple,blockIdx.x, strlenDevice(seq2));
        int counter = 0;

        for (int j = 0; j < strlenDevice(seq2); j++) // generate mutant
        {
            if (j != myTuple.n && j != myTuple.k) // if not equal to n and k
            {
                seq2Shared[counter] = seq2[j];
                counter++;
            }
        }
    }

    __syncthreads();
    int score = calculateScore(seq1, seq2Shared, w, threadIdx.x);
    atomicMax(&bestScore, score);
    __syncthreads();

    if(bestScore == score)
    {
        bestOffset = threadIdx.x;
    }
    __syncthreads();
    if(threadIdx.x == 0)
    {
        atomicMax(maxScore, bestScore);
    }
    __syncthreads();
    if(threadIdx.x == 0)
    {
        if(bestScore == *maxScore)
        {
            nk->n = myTuple.n;
            nk->k = myTuple.k;
            *offset = bestOffset;
        }
    free(seq2Shared);
    }
}

extern "C" int* getBestMutantCuda(char *seq1, char *seq2, int w[])
{
    int* result = (int*)malloc(sizeof(int) * 3);
    char* seq1Device;
    hipMalloc(&seq1Device, sizeof(char) * strlenDevice(seq1) + 1);
    hipMemcpy(seq1Device, seq1, sizeof(char) * strlenDevice(seq1) + 1, hipMemcpyHostToDevice);
    char* seq2Device;
    int* weightsDevice;
    hipMalloc(&seq2Device, sizeof(char) * strlenDevice(seq2) + 1);
    hipMalloc(&weightsDevice, sizeof(int) * 4);
    hipMemcpy(seq2Device, seq2, sizeof(char) * strlenDevice(seq2) + 1, hipMemcpyHostToDevice);
    hipMemcpy(weightsDevice, w, sizeof(int) * 4, hipMemcpyHostToDevice);
    nkTuple* nkDevice;
    int* offsetDevice;
    nkTuple* nk = (nkTuple*)malloc(sizeof(nkTuple));
    hipMalloc(&nkDevice, sizeof(nkTuple));
    hipMalloc(&offsetDevice, sizeof(int));
    int n = strlen(seq2) - 1;
    int s = n * (1 + n) / 2; // sum of all possible mutants
    int* maxScore;
    hipMalloc(&maxScore,sizeof(int));
    int* helper = (int*)malloc(sizeof(int));
    *helper = 999;
    BestMutantANDOffset<<<s,strlen(seq1)-strlen(seq2) + 3>>>(nkDevice,seq1Device,seq2Device,weightsDevice,offsetDevice,maxScore);
    hipMemcpy(helper,maxScore,sizeof(int),hipMemcpyDeviceToHost);
    printf("max Score (helper): %d\n",*helper);
    hipMemcpy(result,offsetDevice,sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(nk,nkDevice,sizeof(nkTuple),hipMemcpyDeviceToHost);
    result[2] = nk->k;
    result[1] = nk->n;

    printf("%d ,%d ,%d\n",result[0],result[1],result[2]);
    free(nk);
    
    hipFree(seq1Device);
    hipFree(seq2Device);
    hipFree(weightsDevice);
    hipFree(nkDevice);
    hipFree(maxScore);
    hipFree(offsetDevice);
    return result;
}
