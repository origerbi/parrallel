#include <hip/hip_runtime.h>
#include <limits.h>


__device__ const char *conservativeGroup[] = {"NDEQ", "MILV", "FYW", "NEQK", "QHRK", "HY", "STA", "NHQK", "MILF"};
__device__ const char *semiConservativeGroup[] = {"SAG", "SGND", "NEQHRK", "ATV", "STPA", "NDEQHK", "HFY", "CSA", "STNK", "SNDEQK", "FVLIM"};

typedef struct
{
    int n;
    int k;
}nkTuple;


// calculate string length
__host__ __device__ int strlenDevice(const char *str)
{
    int i = 0;
    while (str[i] != '\0')
    {
        i++;
    }
    return i;
}

__device__ int inString(const char *str, char c)
{
    for (int i = 0; str[i] != '\0'; i++)
    {
        if (str[i] == c)
        {
            return 0;
        }
    }
    return -1;
}

__device__ int compareInGroup(char a, char b, const char *group[], int groupLength)
{
    for (int i = 0; i < 9; i++)
    {
        if (inString(group[i], a) && inString(group[i], b))
            return 0;
    }
    return -1;
}

__device__ char getSign(char a, char b)
{
    if (a == b)
    {
        return '*';
    }
    switch (compareInGroup(a, b, conservativeGroup, 9))
    {
    case 0:
        return ':';
    case -1:
        switch (compareInGroup(a, b, semiConservativeGroup, 11))
        {
        case 0:
            return '.';
        default: // case -1: not in semiConservativeGroup
            return ' ';
        }
    }
    return ' ';
}

__device__ int calculateScore(char *seq1, char *seq2, int w[], int offset)
{
    int starsCount = 0;
    int colonsCount = 0;
    int pointsCount = 0;
    int spacesCount = 0;
    for (int i = 0; i < strlenDevice(seq2); i++)
    {
        switch (getSign(seq1[i], seq2[i + offset]))
        {
        case '*':
            starsCount++;
            break;
        case ':':
            colonsCount++;
            break;
        case '.':
            pointsCount++;
            break;
        case ' ':
            spacesCount++;
            break;
        }
    }
    return w[0] * starsCount - w[1] * colonsCount - w[2] * pointsCount - w[3] * spacesCount;
}

__device__ nkTuple getNKFromNumber(int number, int lengthSeq2)
{
    nkTuple tuple;
    tuple.n = 0;
    tuple.k = 1;
    for(int i = 0; i < number; i++)
    {
        if (tuple.k < lengthSeq2 - 1) // increment k
            tuple.k++;
        else // increment n and reset k
        {
            tuple.n++;
            tuple.k = tuple.n + 1;
        }
    }
    return tuple;
}

__global__ void BestMutantANDOffset(nkTuple* nk, char* seq1, char* seq2, int* w, int* offset)           // each block represent mutant, each thread represent offset.
{
    // cuda style
    int result = 0;
    int maxScore = INT_MIN;
    __shared__ int bestOffset;
    __shared__ int bestScore;
    __shared__ char* seq2Shared;
    __shared__ nkTuple myTuple;
    if(threadIdx.x == 0)
    {
        bestOffset = 0;
        bestScore = INT_MIN;
        seq2Shared = (char*)malloc(sizeof(char) * strlenDevice(seq2));
        myTuple = getNKFromNumber(blockIdx.x, strlenDevice(seq2));
        int counter = 0;
        for (int j = 0; j < strlenDevice(seq2); j++) // generate mutant
        {
            if (j != myTuple.n && j != myTuple.k) // if not equal to n and k
            {
                seq2Shared[counter] = seq2[j];
                counter++;
            }
        }
    }
    __syncthreads();
    int score = calculateScore(seq1, seq2Shared, w, threadIdx.x);
    atomicMax(&bestScore, score);
    __syncthreads();
    if(bestScore == score)
    {
        bestOffset = threadIdx.x;
    }
    __syncthreads();
    if(threadIdx.x == 0)
    {
        atomicMax(&maxScore, bestScore);

    }
    __syncthreads();
    if(threadIdx.x == 0)
    {
        if(bestScore == maxScore)
        {
            nk->n = myTuple.n;
            nk->k = myTuple.k;
            result = bestOffset;
        }
    }
    free(seq2Shared);
    *offset = result;
}

extern "C" int* getBestMutantCuda(char *seq1, char *seq2, int w[])
{
    int* result = (int*)malloc(sizeof(int) * 3);
    char* seq1Device;
    hipMalloc(&seq1Device, sizeof(char) * strlenDevice(seq1));
    hipMemcpy(seq1Device, seq1, sizeof(char) * strlenDevice(seq1), hipMemcpyHostToDevice);
    char* seq2Device;
    int* weightsDevice;
    hipMalloc(&seq2Device, sizeof(char) * strlenDevice(seq2));
    hipMalloc(&weightsDevice, sizeof(int) * 4);
    hipMemcpy(seq2Device, seq2, sizeof(char) * strlenDevice(seq2), hipMemcpyHostToDevice);
    hipMemcpy(weightsDevice, w, sizeof(int) * 4, hipMemcpyHostToDevice);
    nkTuple* nkDevice;
    int* offsetDevice;
    nkTuple* nk = (nkTuple*)malloc(sizeof(nkTuple));
    hipMalloc(&nkDevice, sizeof(nkTuple));
    hipMalloc(&offsetDevice, sizeof(int));
    int n = strlen(seq2) - 1;
    int s = n * (1 + n) / 2; // sum of all possible mutants
    BestMutantANDOffset<<<s,strlen(seq2)-1>>>(nkDevice,seq1Device,seq2Device,weightsDevice,offsetDevice);
    hipMemcpy(result,offsetDevice,sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(nk,nkDevice,sizeof(nkTuple),hipMemcpyDeviceToHost);
    result[2] = nk->k;
    result[1] = nk->n;
    free(nk);
    hipFree(seq1Device);
    hipFree(seq2Device);
    hipFree(weightsDevice);
    hipFree(nkDevice);
    hipFree(offsetDevice);
    return result;
}
